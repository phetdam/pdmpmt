#include "hip/hip_runtime.h"
/**
 * @file curand_mcpi.cu
 * @author Derek Huang
 * @brief CUDA C++ program computing pi using Monte Carlo with cuRAND
 */

#include <cstdlib>
#include <iostream>
#include <utility>

#include <hiprand.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/reduce.h>
#include <thrust/version.h>

#include "pdmpmt/common.h"
#include "pdmpmt/cuda_runtime.hh"

namespace {

/**
 * Return the `hiprandStatus_t` identifier string for the given error status.
 *
 * This covers all the cuRAND status values for cuRAND 10.3.3.
 *
 * @param err cuRAND error status
 */
constexpr auto curand_strerror(hiprandStatus err) noexcept
{
  switch (err) {
#define ERROR_CASE(x) case x: return #x;
  ERROR_CASE(HIPRAND_STATUS_SUCCESS)
  ERROR_CASE(HIPRAND_STATUS_VERSION_MISMATCH)
  ERROR_CASE(HIPRAND_STATUS_NOT_INITIALIZED)
  ERROR_CASE(HIPRAND_STATUS_ALLOCATION_FAILED)
  ERROR_CASE(HIPRAND_STATUS_TYPE_ERROR)
  ERROR_CASE(HIPRAND_STATUS_OUT_OF_RANGE)
  ERROR_CASE(HIPRAND_STATUS_LENGTH_NOT_MULTIPLE)
  ERROR_CASE(HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED)
  ERROR_CASE(HIPRAND_STATUS_LAUNCH_FAILURE)
  ERROR_CASE(HIPRAND_STATUS_PREEXISTING_FAILURE)
  ERROR_CASE(HIPRAND_STATUS_INITIALIZATION_FAILED)
  ERROR_CASE(HIPRAND_STATUS_ARCH_MISMATCH)
  ERROR_CASE(HIPRAND_STATUS_INTERNAL_ERROR)
  default:
    return "(unknown)";
#undef ERROR_CASE
  }
}

/**
 * cuRNAD error handler type.
 */
struct curand_error_handler_type {};

/**
 * Print an error message and exit if a cuRAND status is an error status.
 */
void operator<<(curand_error_handler_type /*handler*/, hiprandStatus_t status)
{
  if (status == HIPRAND_STATUS_SUCCESS)
    return;
  // handle error
  std::cerr << "cuRAND error: " << curand_strerror(status) << std::endl;
  std::exit(EXIT_FAILURE);
}

/**
 * Error handler global for cuRAND functions.
 */
constexpr curand_error_handler_type curand_check;

/**
 * cuRAND generator class.
 *
 * This provides a scoping mechanism for managing a cuRAND generator.
 */
class curand_generator {
public:
  /**
   * Default ctor.
   *
   * The created generator will be a MT19937 Mersenne Twister generator.
   */
  curand_generator() : curand_generator{HIPRAND_RNG_PSEUDO_MT19937} {}

  /**
   * Ctor.
   *
   * Create a generator of the specified type.
   *
   * @param type cuRAND generator type
   */
  curand_generator(hiprandRngType_t type) : type_{type}
  {
    curand_check << hiprandCreateGenerator(&gen_, type);
  }

  /**
   * Deleted copy ctor.
   */
  curand_generator(const curand_generator&) = delete;

  /**
   * Move ctor.
   */
  curand_generator(curand_generator&& other) noexcept
  {
    from(std::move(other));
  }

  /**
   * Dtor.
   */
  ~curand_generator()
  {
    destroy();
  }

  /**
   * Move assignment operator.
   */
  auto& operator=(curand_generator&& other) noexcept
  {
    destroy();
    from(std::move(other));
    return *this;
  }

  /**
   * Enable implicit conversion to `hiprandGenerator_t` for cuRAND interop.
   *
   * This can also be used to indicate if the generator is valid or not.
   */
  operator hiprandGenerator_t() const noexcept
  {
    return gen_;
  }

private:
  hiprandRngType_t type_;
  hiprandGenerator_t gen_{};

  /**
   * Move-initialize from another `curand_generator`.
   *
   * On completion the other `curand_generator` will have a null generator.
   */
  void from(curand_generator&& other) noexcept
  {
    gen_ = other.gen_;
    other.gen_ = nullptr;
  }

  /**
   * Destroy the cuRAND generator.
   *
   * This will never fail since the cuRAND generator wlll have been created.
   */
  void destroy() noexcept
  {
    if (gen_)
      hiprandDestroyGenerator(gen_);
  }
};

/**
 * 1D span for CUDA code.
 */
template <typename T>
class span {
public:
  /**
   * Ctor.
   *
   * Creates an empty span.
   */
  PDMPMT_XPU_FUNC
  span() noexcept = default;

  /**
   * Ctor.
   *
   * @param data Data buffer
   * @param size Buffer element count.
   */
  PDMPMT_XPU_FUNC
  span(T* data, std::size_t size) noexcept : data_{data}, size_{size} {}

  /**
   * Ctor.
   *
   * Create a span from a Thrust device vector.
   *
   * @param vec Thrust device vector
   */
  span(thrust::device_vector<T>& vec) noexcept
    : data_{vec.data().get()}, size_{vec.size()}
  {}

  /**
   * Return the data pointer.
   */
  PDMPMT_XPU_FUNC
  auto data() const noexcept { return data_; }

  /**
   * Return the element count.
   */
  PDMPMT_XPU_FUNC
  auto size() const noexcept { return size_; }

  /**
   * Return the `i`th element in the span.
   */
  PDMPMT_XPU_FUNC
  auto& operator[](std::size_t i) noexcept
  {
    return data_[i];
  }

  /**
   * Return the `i`th element in the span.
   */
  PDMPMT_XPU_FUNC
  const auto& operator[](std::size_t i) const noexcept
  {
    return data_[i];
  }

  /**
   * Return an iterator to the first element in the span.
   */
  PDMPMT_XPU_FUNC
  auto begin() const noexcept
  {
    return data_;
  }

  /**
   * Return an iterator one past the last element in the span.
   */
  PDMPMT_XPU_FUNC
  auto end() const noexcept
  {
    return data_;
  }

private:
  T* data_{};
  std::size_t size_{};
};

/**
 * Check how many points fall within the quarter-unit circle.
 *
 * All dimensions are assumed to be 1D.
 *
 * @tparam T Floating type
 *
 * @param ns Number of samples
 * @param xs x-axis samples in (0, 1]
 * @param ys y-axis sampels in (0, 1]
 * @param cts Per-thread counts of points within the quarter-unit circle
 */
template <typename T>
__global__ void
unit_circle_check(const span<T> xs, const span<T> ys, span<unsigned> cts)
{
  // get number of samples and number of threads
  // note: assumes xs and ys have the same size
  auto ns = xs.size();
  auto nt = cts.size();
  // get the thread index
  auto ti = blockIdx.x * blockDim.x + threadIdx.x;
  // compute starting index
  // determine work unit size (last thread gets the remaining work)
  auto work_size = ns / nt;
  if (ti == nt - 1)
    work_size += ns % nt;
  // starting index offset
  auto offset = ns / nt * ti;
  // count number of points in the quarter-unit circle
  unsigned n_in = 0u;
  for (auto i = offset; i < offset + work_size; i++)
    n_in += (xs[i] * xs[i] + ys[i] * ys[i] <= 1);
  // update per-thread count vector
  cts[ti] = n_in;
}

}  // namespace

int main()
{
  // cuRAND version info
  std::cout << "cuRAND version: " <<
    CURAND_VER_MAJOR << "." << CURAND_VER_MINOR << "." << CURAND_VER_PATCH <<
    std::endl;
  // seed + sample count
  constexpr auto seed = 8888u;
  constexpr auto n_samples = 10'000'000u;
  // number of threads per block + number of blocks + total thread count
  constexpr auto n_block_threads = 256u;
  constexpr auto n_blocks = 2048u;
  constexpr auto n_threads = n_block_threads * n_blocks;
  // create cuRAND Mersenne Twister generator using best possible memory order
  curand_generator gen{HIPRAND_RNG_PSEUDO_MT19937};
  curand_check << hiprandSetPseudoRandomGeneratorSeed(gen, seed);
  curand_check << hiprandSetGeneratorOrdering(gen, HIPRAND_ORDERING_PSEUDO_BEST);
  // create device vectors for x and y point coordinates in (0, 1]
  thrust::device_vector<float> xs(n_samples);
  thrust::device_vector<float> ys(n_samples);
  // create device vector to hold per-thread counts of points in circle
  thrust::device_vector<unsigned> cts(n_threads);
  // create spans for device memory from the vectors
  span xs_view{xs};
  span ys_view{ys};
  span cts_view{cts};
  // fill device vectors with uniform values + block until completion
  curand_check << hiprandGenerateUniform(gen, xs_view.data(), xs_view.size());
  curand_check << hiprandGenerateUniform(gen, ys_view.data(), ys_view.size());
  hipDeviceSynchronize();
  PDMPMT_CUDA_THROW_IF_ERROR();
  // launch kernel to count number of points in the quarter-unit circle
  unit_circle_check<<<n_blocks, n_block_threads>>>(xs_view, ys_view, cts_view);
  hipDeviceSynchronize();
  PDMPMT_CUDA_THROW_IF_ERROR();
  // accumulate all values to get final count + compute pi
  auto n_in = thrust::reduce(thrust::device, cts.begin(), cts.end(), 0u);
  auto pi = (4. * n_in) / n_samples;
  // print
  std::cout << "pi (n_threads=" << n_threads << ", n_samples=" << n_samples <<
    ", seed=" << seed << "): " << pi << std::endl;
  return EXIT_SUCCESS;
}

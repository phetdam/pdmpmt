/**
 * @file mcpi.cu
 * @author Derek Huang
 * @brief CUDA C++ implementation for estimating pi with Monte Carlo
 * @copyright MIT License
 */

#include <cstdlib>
#include <iostream>

#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/sequence.h>
#include <thrust/version.h>

#include "pdmpmt/type_traits.hh"

namespace {

/**
 * Exit with error if the CUDA error code indicates failure.
 *
 * @param err CUDA error code
 */
void cuda_check(hipError_t err)
{
  // success
  if (err == hipSuccess)
    return;
  // failure
  std::cerr << "CUDA error: " << hipGetErrorName(err) << ": " <<
    hipGetErrorString(err) << std::endl;
  std::exit(EXIT_FAILURE);
}

/**
 * Exit with error if the last CUDA runtime API call failed.
 */
void cuda_check()
{
  cuda_check(hipGetLastError());
}

/**
 * Insert the contents of a device vector into the stream.
 *
 * @tparam T Streamable type
 * @tparam A Allocator
 *
 * @param out Output stream
 * @param vec Device vector to write
 */
template <typename T, typename A, typename = pdmpmt::ostreamable_t<T>>
auto& operator<<(std::ostream& out, const thrust::device_vector<T, A>& vec)
{
  out << '[';
  for (auto it = vec.begin(); it != vec.end(); it++) {
    if (it != vec.begin())
      out << ", ";
    out << *it;
  }
  return out << ']';
}

}  // namespace

int main()
{
  // print some version info
  int dr_ver, rt_ver;
  hipDriverGetVersion(&dr_ver);
  cuda_check();
  std::cout << "CUDA driver version: " <<
    (dr_ver / 1000) << "." << (dr_ver % 100 / 10) << std::endl;
  hipRuntimeGetVersion(&rt_ver);
  cuda_check();
  std::cout << "CUDA runtime version: " <<
    (rt_ver / 1000) << "." << (rt_ver % 100 / 10) << std::endl;
  // print the Thrust version
  std::cout << "Thrust version: " << THRUST_MAJOR_VERSION << "." <<
    THRUST_MINOR_VERSION << "." << THRUST_SUBMINOR_VERSION << std::endl;
  // create device vector ascending sequence
  thrust::device_vector<double> values(4);
  thrust::sequence(values.begin(), values.end());
  // print and reduce (accumulate)
  std::cout << "values are " << values << std::endl;
  auto sum = thrust::reduce(values.begin(), values.end());
  std::cout << "sum is " << sum << std::endl;
  return EXIT_SUCCESS;
}

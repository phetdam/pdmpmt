/**
 * @file mcpi.cu
 * @author Derek Huang
 * @brief CUDA C++ implementation for estimating pi with Monte Carlo
 * @copyright MIT License
 */

#include <cstdlib>
#include <iostream>

#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/sequence.h>
#include <thrust/version.h>

#include "pdmpmt/cuda_runtime.hh"
#include "pdmpmt/type_traits.hh"

namespace {

/**
 * Insert the contents of a device vector into the stream.
 *
 * @tparam T Streamable type
 * @tparam A Allocator
 *
 * @param out Output stream
 * @param vec Device vector to write
 */
template <typename T, typename A, typename = pdmpmt::ostreamable_t<T>>
auto& operator<<(std::ostream& out, const thrust::device_vector<T, A>& vec)
{
  out << '[';
  for (auto it = vec.begin(); it != vec.end(); it++) {
    if (it != vec.begin())
      out << ", ";
    out << *it;
  }
  return out << ']';
}

}  // namespace

int main()
{
  // print driver and runtime versions
  std::cout <<
    "CUDA driver version: " << pdmpmt::cuda_driver_version() << '\n' <<
    "CUDA runtime version: " << pdmpmt::cuda_runtime_version() << std::endl;
  // print the Thrust version
  std::cout << "Thrust version: " << THRUST_MAJOR_VERSION << "." <<
    THRUST_MINOR_VERSION << "." << THRUST_SUBMINOR_VERSION << std::endl;
  // create device vector ascending sequence
  thrust::device_vector<double> values(4);
  thrust::sequence(values.begin(), values.end());
  // print and reduce (accumulate)
  std::cout << "values are " << values << std::endl;
  auto sum = thrust::reduce(values.begin(), values.end());
  std::cout << "sum is " << sum << std::endl;
  return EXIT_SUCCESS;
}
